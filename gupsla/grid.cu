#include "hip/hip_runtime.h"
#include "grid.h"

#include <stdio.h>
#include <time.h>
#include <math.h>
#define CELLS_SIZE grid->Tables * grid->Rows * grid->Columns * sizeof(byte)


char grid_chars[255]{ '_', 'X' };

bool init = false;
hipDeviceProp_t gpu_info;
int block_width;
dim3 threads;
size_t shared_memory_size;

Grid * Grid_Create(int tables, int rows, int columns)
{
	Grid * grid = alloc(1, Grid);

	grid->Tables = tables;
	grid->Rows = rows;
	grid->Columns = columns;
	grid->Cells = alloc(tables * rows * columns, byte);

	if (!init)
	{
		srand((unsigned int)time(NULL));
		gpuOk(hipGetDeviceProperties(&gpu_info, 0));
		init = true;

		block_width = (int)sqrt(gpu_info.maxThreadsPerBlock) - 2;
		int sh_width = (int)(sqrt(gpu_info.sharedMemPerBlock + 4) - 2);
		if (sh_width < block_width) block_width = sh_width;

		threads = dim3(block_width, block_width, 1);
		shared_memory_size = block_width * (block_width + 4) * sizeof(byte);
	}
	gpuOk(hipMalloc(&(grid->dev_Cells), CELLS_SIZE));

	grid->Step = &Grid_D_GoLStep;

	return grid;
}

void Grid_Destroy(Grid ** grid_ptr)
{
	Grid * grid = *grid_ptr;

	free(grid->Cells);
	hipFree(grid->dev_Cells);
	free(grid);

	grid_ptr = NULL;
}

void Grid_Randomize(Grid * grid)
{
	for (int t = 0; t < grid->Tables; t++)
		for (int r = 0; r < grid->Rows; r++)
			for (int c = 0; c < grid->Columns; c++)
				grid->Cells[TRCTC(t, r, c, grid->Rows, grid->Columns)] = rand() % 2;
}

void Grid_Print(Grid * grid, bool draw_outline)
{
	if (draw_outline)
	{
		for (int c = 0; c < grid->Columns + 2; c++)
			putchar('#');
		putchar('\n');
	}

	for (int r = 0; r < grid->Rows; r++)
	{
		if (draw_outline) putchar('#');
		for (int c = 0; c < grid->Columns; c++)
		{
			putchar(grid_chars[grid->Cells[RCC(r, c, grid->Columns)]]);
		}
		if (draw_outline) putchar('#');
		putchar('\n');
	}

	if (draw_outline)
	{
		for (int c = 0; c < grid->Columns + 2; c++)
			putchar('#');
		putchar('\n');
	}
}

void Grid_Upload(Grid * grid)
{
	gpuOk(hipMemcpy(grid->dev_Cells, grid->Cells, CELLS_SIZE, hipMemcpyHostToDevice));
}

void Grid_Download(Grid * grid)
{
	gpuOk(hipMemcpy(grid->Cells, grid->dev_Cells, CELLS_SIZE, hipMemcpyDeviceToHost));
}

#define devide_ceiling(a, b) (int)ceil((double)(a) / (b))
void Grid_Step(Grid * grid)
{
	dim3 blocks(devide_ceiling(grid->Columns, block_width), devide_ceiling(grid->Rows, block_width));
	dim3 grid_size(grid->Columns, grid->Rows, grid->Tables);

	printf("SIZE   : %d x %d x %d\n", grid_size.x, grid_size.y, grid_size.z);
	printf("BLOCKS : %d x %d x %d\n", blocks.x, blocks.y, blocks.z);
	printf("THREADS: %d x %d x %d\n", threads.x, threads.y, threads.z);
	printf("SHARED : %g kB\n", ceil(shared_memory_size / 1024.0));
	
	grid->Step DEV3(blocks, threads, shared_memory_size) (grid->dev_Cells, grid_size);
}

__global__ void Grid_D_GoLStep(byte * device_grid, dim3 size)
{
	int neighbours = 0;
	bool state = false;

	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	if (x >= size.x || y >= size.y)
	{
		syncthreads();
		return;
	}

	int i = XYW(x, y, size.x);

	state = device_grid[i] == 1;
	bool space_up = threadIdx.y > 0;
	bool space_down = threadIdx.y < size.y - 1;

	if (threadIdx.x > 0)
	{
		if (space_up) neighbours += device_grid[i - 1 - size.x];
		neighbours += device_grid[i - 1];
		if (space_down) neighbours += device_grid[i - 1 + size.x];
	}

	// if (true)
	{
		if (space_up) neighbours += device_grid[i - size.x];
		if (space_down) neighbours += device_grid[i + size.x];
	}

	if (threadIdx.x < size.x)
	{
		if (space_up) neighbours += device_grid[i + 1 - size.x];
		neighbours += device_grid[i + 1];
		if (space_down) neighbours += device_grid[i + 1 + size.x];
	}

	syncthreads();

	device_grid[i] = neighbours == 3 || (state && neighbours == 2);

}