#include "hip/hip_runtime.h"
#include "grid.h"

#include <stdio.h>
#include <time.h>
#include <math.h>
#define CELLS_SIZE grid->Tables * grid->Rows * grid->Columns * sizeof(byte)
#define BOOL_SIZE sizeof(bool)
#define SetIdleTrue()  grid->Idle = true ; gpuOk(hipMemcpy(grid->dev_Idle, &True , BOOL_SIZE, hipMemcpyHostToDevice))
#define SetIdleFalse() grid->Idle = false; gpuOk(hipMemcpy(grid->dev_Idle, &False, BOOL_SIZE, hipMemcpyHostToDevice))
#define UpdateIdle()   gpuOk(hipMemcpy(&(grid->Idle), grid->dev_Idle, BOOL_SIZE, hipMemcpyDeviceToHost))

bool False = false;
bool True  = true;

char grid_chars[255]{ '_', 'X' };

bool init = false;
hipDeviceProp_t gpu_info;
int block_width;
dim3 threads;
size_t shared_memory_size;

Grid * Grid_Create(int tables, int rows, int columns)
{
	Grid * grid = alloc(1, Grid);

	grid->Tables = tables;
	grid->Rows = rows;
	grid->Columns = columns;
	grid->Cells = alloc(tables * rows * columns, byte);

	if (!init)
	{
		srand((unsigned int)time(NULL));
		gpuOk(hipGetDeviceProperties(&gpu_info, 0));
		init = true;

		block_width = (int)sqrt(gpu_info.maxThreadsPerBlock) - 2;
		int sh_width = (int)(sqrt(gpu_info.sharedMemPerBlock + 4) - 2);
		if (sh_width < block_width) block_width = sh_width;

		threads = dim3(block_width, block_width, 1);
		shared_memory_size = tables * (block_width + 2) * (block_width + 2) * sizeof(byte);
	}
	gpuOk(hipMalloc(&(grid->dev_Cells), CELLS_SIZE));

	grid->kernel_Step = &Grid_D_GoLStep;
	gpuOk(hipMalloc(&(grid->dev_Idle), BOOL_SIZE));
	SetIdleFalse();

	return grid;
}

void Grid_Destroy(Grid ** grid_ptr)
{
	Grid * grid = *grid_ptr;

	free(grid->Cells);
	gpuOk(hipFree(grid->dev_Cells));
	free(grid);

	gpuOk(hipFree(grid->dev_Idle));

	grid_ptr = NULL;
}

void Grid_Randomize(Grid * grid)
{
	for (int t = 0; t < grid->Tables; t++)
		for (int r = 0; r < grid->Rows; r++)
			for (int c = 0; c < grid->Columns; c++)
				grid->Cells[TRCTC(t, r, c, grid->Rows, grid->Columns)] = rand() % 2;
}

void Grid_Print(Grid * grid, bool draw_outline)
{
	if (draw_outline)
	{
		for (int c = 0; c < grid->Columns + 2; c++)
			putchar('#');
		putchar('\n');
	}

	for (int r = 0; r < grid->Rows; r++)
	{
		if (draw_outline) putchar('#');
		for (int c = 0; c < grid->Columns; c++)
		{
			putchar(grid_chars[grid->Cells[RCC(r, c, grid->Columns)]]);
		}
		if (draw_outline) putchar('#');
		putchar('\n');
	}

	if (draw_outline)
	{
		for (int c = 0; c < grid->Columns + 2; c++)
			putchar('#');
		putchar('\n');
	}
}

void Grid_Upload(Grid * grid)
{
	gpuOk(hipMemcpy(grid->dev_Cells, grid->Cells, CELLS_SIZE, hipMemcpyHostToDevice));
}

void Grid_Download(Grid * grid)
{
	gpuOk(hipMemcpy(grid->Cells, grid->dev_Cells, CELLS_SIZE, hipMemcpyDeviceToHost));
}

#define devide_ceiling(a, b) (int)ceil((double)(a) / (b))
void Grid_Step(Grid * grid)
{
	dim3 blocks(devide_ceiling(grid->Columns, block_width), devide_ceiling(grid->Rows, block_width));
	dim3 grid_size(grid->Columns, grid->Rows, grid->Tables);

	printf("SIZE   : %d x %d x %d\n", grid_size.x, grid_size.y, grid_size.z);
	printf("BLOCKS : %d x %d x %d\n", blocks.x, blocks.y, blocks.z);
	printf("THREADS: %d x %d x %d\n", threads.x, threads.y, threads.z);
	printf("SHARED : %g kB\n", ceil(shared_memory_size / 1024.0));
	
	SetIdleTrue();
	grid->kernel_Step DEV3(blocks, threads, shared_memory_size) (grid->dev_Cells, grid->dev_Idle, grid_size);
	UpdateIdle();
}

__global__ void Grid_D_GoLStep(byte * device_grid, bool * device_idle, dim3 size)
{
	#pragma region index generation
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	if (x >= size.x || y >= size.y)
	{
		syncthreads();
		return;
	}

	int global = XYW(x, y, size.x);
	int blocki = XYW(blockIdx.x, blockIdx.y, blockDim.x);
	int shardi = blocki + blockDim.x + 1;
	#pragma endregion

	#pragma region allocate shared memory
	extern __shared__ byte sha_block[];
	sha_block[shardi] = device_grid[global];

	
	syncthreads();
	#pragma endregion

	int neighbours = 0;
	bool state = false;


	state = device_grid[global] == 1;
	bool space_up = threadIdx.y > 0;
	bool space_down = threadIdx.y < size.y - 1;

	if (threadIdx.x > 0)
	{
		if (space_up) neighbours += device_grid[global - 1 - size.x];
		neighbours += device_grid[global - 1];
		if (space_down) neighbours += device_grid[global - 1 + size.x];
	}

	// if (true)
	{
		if (space_up) neighbours += device_grid[global - size.x];
		if (space_down) neighbours += device_grid[global + size.x];
	}

	if (threadIdx.x < size.x)
	{
		if (space_up) neighbours += device_grid[global + 1 - size.x];
		neighbours += device_grid[global + 1];
		if (space_down) neighbours += device_grid[global + 1 + size.x];
	}

	syncthreads();
	byte value = neighbours == 3 || (state && neighbours == 2);
	if (device_grid[global] != value)
	{
		device_grid[global] = value;
		*device_idle = false;
	}

}