#include "cuda_ext.cuh"
/* LNK4221
inline hipError_t gpuAssert(hipError_t code, const char * file, int line)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "CUDA:\n*****\n%s\n%s\nat %s : %d\n", hipGetErrorName(code), hipGetErrorString(code), file, line);
		getchar(); exit(-1);
	}
	return code;
}
// */